#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include "kernel.h"

#define TPB 64

__device__
float distance(float x1, float x2) {
  return sqrt((x2-x1)*(x2-x1));
}

__global__
void distanceKernel(float *d_out, float *d_in, float ref) {
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  const float x = d_in[i];
  d_out[i] = distance(x, ref);
  // printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

float scale(int i, int n) {
  return ((float)i)/(n-1);
}

void distanceArray(float ref, int len) {
    int dev = 0;
        hipDeviceProp_t devProp;
            hipGetDeviceProperties(&devProp, dev);
                std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
                    std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
                        std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
                            std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
                                std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
                                    std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;


  float *in = NULL;
  float *out = NULL;

  hipMallocManaged(&in, len * sizeof(float));
  hipMallocManaged(&out, len * sizeof(float));

  for (int i = 0; i < len; i++) {
    in[i] = scale(i, len);
  }

  distanceKernel<<<len/TPB, TPB>>> (out, in, ref);
  hipDeviceSynchronize();

  hipFree(in);
  hipFree(out);
}
